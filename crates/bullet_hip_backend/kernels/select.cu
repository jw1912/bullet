#include "hip/hip_runtime.h"
#include "util.cu"
#ifdef __HIP_PLATFORM_AMD__
#include <hip/hip_runtime.h>
#endif

__global__ void selectKernel(
    const size_t batch_size,
    const int32_t input_batched,
    const size_t input_size,
    const size_t output_size,
    const int32_t* buckets,
    const float* in,
    float* out)
{
    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= batch_size * output_size)
        return;

    const size_t idxInBatch = tid / output_size;
    const size_t idxInOutput = tid % output_size;

    const size_t thisBucket = static_cast<size_t>(buckets[idxInBatch]);

    const float thisInput = in[output_size * thisBucket + idxInOutput + (input_batched ? input_size * idxInBatch : 0)];
    out[output_size * idxInBatch + idxInOutput] = thisInput;
}

__global__ void selectBackpropKernel(
    const size_t batch_size,
    const int32_t input_grad_batched,
    const size_t input_size,
    const size_t output_size,
    const int32_t* buckets,
    const float* output_grad,
    float* input_grad)
{
    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= batch_size * output_size)
        return;

    const size_t idxInBatch = tid / output_size;
    const size_t idxInOutput = tid % output_size;

    const size_t thisBucket = static_cast<size_t>(buckets[idxInBatch]);

    const float thisOutputGrad = output_grad[output_size * idxInBatch + idxInOutput];
    float* thisInputGrad = input_grad + output_size * thisBucket + idxInOutput;

    if (input_grad_batched)
    {
        thisInputGrad[input_size * idxInBatch] += thisOutputGrad;
    }
    else
    {
        atomicAdd(thisInputGrad, thisOutputGrad);
    }
}

extern "C" void selectForward(
    const size_t batch_size,
    const int32_t input_batched,
    const size_t input_size,
    const size_t output_size,
    const int32_t* buckets,
    const float* in,
    float* out)
{
    const size_t blocks = (batch_size * output_size + threadsPerBlock - 1) / threadsPerBlock;

    selectKernel<<<blocks, threadsPerBlock>>>(
        batch_size,
        input_batched,
        input_size,
        output_size,
        buckets,
        in,
        out
    );
}

extern "C" void selectBackprop(
    const size_t batch_size,
    const int32_t input_grad_batched,
    const size_t input_size,
    const size_t output_size,
    const int32_t* buckets,
    const float* output_grad,
    float* input_grad)
{
    const size_t blocks = (batch_size * output_size + threadsPerBlock - 1) / threadsPerBlock;

    selectBackpropKernel<<<blocks, threadsPerBlock>>>(
        batch_size,
        input_grad_batched,
        input_size,
        output_size,
        buckets,
        output_grad,
        input_grad
    );
}
